#include "hip/hip_runtime.h"
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"

// ============================================================================================
// Types
// ============================================================================================

struct OriginalAir {
    int width;               // number of columns
    int height;              // number of rows (Ha)
    const Fp* buffer;        // column-major base: col*height + row
    int row_block_size;      // stride between used rows
    int substitutions_offset;// offset in d_subs
    int substitutions_length;// count in d_subs for this AIR
};

struct Subst {
    int col;      // source column within this AIR
    int row;      // base row offset within the row-block
    int apc_col;  // destination APC column
};

// ============================================================================================
// Kernel: one block per OriginalAir; each warp handles one substitution (APC column).
// ============================================================================================

__global__ void apc_tracegen_kernel(
    Fp* __restrict__ d_output,                         // [output_height * output_width], column-major
    const OriginalAir* __restrict__ d_original_airs,   // metadata per AIR
    const Subst* __restrict__ d_subs,                  // all substitutions
    size_t output_height,                              // H_out
    int num_apc_calls                                  // number of APC calls
) {
    const int air_id = blockIdx.x;
    const OriginalAir air = d_original_airs[air_id];

    const Fp* __restrict__ src_base = air.buffer;
    const int Ha  = air.height;
    const int RBS = air.row_block_size;

    const int lane  = threadIdx.x & 31;     // 0..31
    const int warp  = threadIdx.x >> 5;     // warp index in block
    const int warps_per_block = blockDim.x >> 5;

    // Process this AIR's substitutions in batches of warps_per_block
    for (int base = 0; base < air.substitutions_length; base += warps_per_block) {
        const int rel = base + warp;
        if (rel >= air.substitutions_length) break;

        const Subst sub = d_subs[air.substitutions_offset + rel];

        // Column bases (column-major)
        const size_t dst_col_base = (size_t)sub.apc_col * (size_t)output_height;
        const size_t src_col_base = (size_t)sub.col     * (size_t)Ha;

        // Each lane writes rows lane, lane+32, lane+64, ... (coalesced per warp)
        // Loop over full output height; zero-pad rows beyond `num_apc_calls`.
        for (size_t r = (size_t)lane; r < (size_t)output_height; r += 32) {
            if (r < (size_t)num_apc_calls) {
                const size_t src_r = (size_t)sub.row + r * (size_t)RBS;
                if (src_r < (size_t)Ha) {
                    d_output[dst_col_base + r] = src_base[src_col_base + src_r];
                }
            } else {
                d_output[dst_col_base + r] = Fp(0);
            }
        }
        // Warps are independent for different substitutions; no syncthreads needed here.
    }
}

// ============================================================================================
// Derived columns application: warp-per-derived-column, lane-per-row
// ============================================================================================

__global__ void apc_apply_derived_kernel(
    Fp* __restrict__ d_output,   // APC trace (column-major)
    size_t H,                    // rows (height)
    int num_apc_calls,           // number of valid rows
    const int* __restrict__ d_cols,  // APC column indices for derived constants
    const Fp* __restrict__ d_vals,   // corresponding constant values
    size_t n_cols
) {
    const int warps_per_block = (blockDim.x >> 5);
    const int warp = (threadIdx.x >> 5);
    const int lane = (threadIdx.x & 31);

    for (int base = (int)blockIdx.x * warps_per_block; base < (int)n_cols; base += (int)gridDim.x * warps_per_block) {
        const int i = base + warp;
        if (i >= (int)n_cols) return;
        const int col = d_cols[i];
        const Fp val = d_vals[i];
        const size_t col_base = (size_t)col * H;
        for (size_t r = (size_t)lane; r < H; r += 32) {
            d_output[col_base + r] = (r < (size_t)num_apc_calls) ? val : Fp(0);
        }
    }
}

extern "C" int _apc_apply_derived(
    Fp*                d_output,
    size_t             output_height,
    int                num_apc_calls,
    const int*         d_cols,
    const Fp*          d_values,
    size_t             n_cols
) {
    if (n_cols == 0) return 0;
    const int block_x = 128; // 4 warps per block
    const dim3 block(block_x, 1, 1);
    unsigned g = (unsigned)((n_cols + 3) / 4);
    if (g == 0u) g = 1u;
    const dim3 grid(g, 1, 1);
    apc_apply_derived_kernel<<<grid, block>>>(
        d_output, output_height, num_apc_calls, d_cols, d_values, n_cols
    );
    return (int)hipGetLastError();
}

// ============================================================================================
// Host launcher wrapper — callable from Rust FFI or cudarc
// ============================================================================================

extern "C" int _apc_tracegen(
    Fp*                      d_output,          // [output_height * output_width], column-major
    size_t                   output_height,     // H_out
    const OriginalAir*       d_original_airs,   // device array, length = n_airs
    size_t                   n_airs,            // one block per AIR
    const Subst*             d_subs,            // device array of all substitutions
    int                      num_apc_calls      // number of APC calls
) {
    assert((output_height & (output_height - 1)) == 0);  // power-of-two height check

    const int block_x = 32;
    const dim3 block(block_x, 1, 1);
    const dim3 grid((unsigned int)n_airs, 1, 1);

    apc_tracegen_kernel<<<grid, block>>>(
        d_output, d_original_airs, d_subs, output_height, num_apc_calls
    );
    return (int)hipGetLastError();
}