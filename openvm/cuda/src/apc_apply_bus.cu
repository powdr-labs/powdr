#include "hip/hip_runtime.h"
#include <stdint.h>
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"
#include "primitives/histogram.cuh"

extern "C" {
typedef struct {
  uint32_t id;
  uint32_t num_args;
  uint32_t mult_off;
  uint32_t mult_len;
  uint32_t args_index_off;
} DevInteraction;

typedef struct {
  uint32_t off;
  uint32_t len;
} DevArgSpan;
}

enum OpCode : uint32_t { OP_PUSH_APC = 0, OP_PUSH_CONST = 1, OP_ADD = 2, OP_SUB = 3, OP_MUL = 4, OP_NEG = 5 };

__device__ __forceinline__ Fp eval_prog(const uint32_t* prog, uint32_t len,
                                        const Fp* __restrict__ col_major,
                                        size_t H, size_t r) {
  Fp stack[16]; int sp = 0; // not sure if enough or what's the typical stack size for our expressions
  for (uint32_t ip = 0; ip < len; ) {
    uint32_t op = prog[ip++];
    switch (op) {
      case OP_PUSH_APC: {
        uint32_t base = prog[ip++];
        stack[sp++] = col_major[base + r];
        break;
      }
      case OP_PUSH_CONST: {
        uint32_t u = prog[ip++];
        stack[sp++] = Fp(u);
        break;
      }
      case OP_ADD: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a + b; break; }
      case OP_SUB: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a - b; break; }
      case OP_MUL: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a * b; break; }
      case OP_NEG: { Fp a = stack[--sp]; stack[sp++] = -a; break; }
    }
  }
  return stack[sp - 1];
}

__global__ void apc_apply_bus_kernel(
  const Fp* __restrict__ d_output, size_t H,
  const DevInteraction* __restrict__ d_interactions, size_t n_interactions,
  const DevArgSpan* __restrict__ d_arg_spans, size_t n_arg_spans,
  const uint32_t* __restrict__ d_bytecode, size_t bc_len,
    int num_apc_calls,
    // bus ids
    uint32_t var_range_bus_id,
    uint32_t tuple2_bus_id,
    uint32_t bitwise_bus_id,
    // histograms and params
    uint32_t* __restrict__ d_var_hist,
    size_t var_num_bins,
    uint32_t* __restrict__ d_tuple2_hist,
    uint32_t tuple2_sz0,
    uint32_t tuple2_sz1,
    uint32_t* __restrict__ d_bitwise_hist,
    uint32_t bitwise_num_bits
) {
  const int warp = (threadIdx.x >> 5);
  const int lane = (threadIdx.x & 31);
  const int warps_per_block = (blockDim.x >> 5);
  for (int base = blockIdx.x * warps_per_block; base < (int)n_interactions; base += gridDim.x * warps_per_block) {
    int i = base + warp;
    if (i >= (int)n_interactions) return;
    DevInteraction intr = d_interactions[i];

    for (int r = lane; r < num_apc_calls; r += 32) {
      const uint32_t* mult_prog = d_bytecode + intr.mult_off;
      Fp mult = eval_prog(mult_prog, intr.mult_len, d_output, H, (size_t)r);
      // Evaluate args and apply based on bus id
      if (intr.id == var_range_bus_id) {
        // expect [value, max_bits]
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 0];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 1];
        Fp v_fp = eval_prog(d_bytecode + s0.off, s0.len, d_output, H, (size_t)r);
        Fp b_fp = eval_prog(d_bytecode + s1.off, s1.len, d_output, H, (size_t)r);
        uint32_t value = v_fp.asUInt32();
        uint32_t max_bits = b_fp.asUInt32();
        lookup::Histogram hist(d_var_hist, (uint32_t)var_num_bins);
        uint32_t idx = (1u << max_bits) + value;
        // apply multiplicity by looping; warp-level dedup in Histogram minimizes contention
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) hist.add_count(idx);
      } else if (intr.id == tuple2_bus_id) {
        // expect [v0, v1]
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 0];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 1];
        Fp v0_fp = eval_prog(d_bytecode + s0.off, s0.len, d_output, H, (size_t)r);
        Fp v1_fp = eval_prog(d_bytecode + s1.off, s1.len, d_output, H, (size_t)r);
        uint32_t v0 = v0_fp.asUInt32();
        uint32_t v1 = v1_fp.asUInt32();
        lookup::Histogram hist(d_tuple2_hist, tuple2_sz0 * tuple2_sz1);
        uint32_t idx = v0 * tuple2_sz1 + v1;
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) hist.add_count(idx);
      } else if (intr.id == bitwise_bus_id) {
        // expect [x, y, x_xor_y, selector]; we only update histogram if selector==range(0) or xor(1)
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 0];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 1];
        DevArgSpan s2 = d_arg_spans[intr.args_index_off + 2];
        DevArgSpan s3 = d_arg_spans[intr.args_index_off + 3];
        Fp x_fp = eval_prog(d_bytecode + s0.off, s0.len, d_output, H, (size_t)r);
        Fp y_fp = eval_prog(d_bytecode + s1.off, s1.len, d_output, H, (size_t)r);
        Fp xy_fp = eval_prog(d_bytecode + s2.off, s2.len, d_output, H, (size_t)r);
        Fp sel_fp = eval_prog(d_bytecode + s3.off, s3.len, d_output, H, (size_t)r);
        uint32_t x = x_fp.asUInt32();
        uint32_t y = y_fp.asUInt32();
        uint32_t xy = xy_fp.asUInt32();
        uint32_t selector = sel_fp.asUInt32();
        BitwiseOperationLookup bl(d_bitwise_hist, bitwise_num_bits);
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) {
          if (selector == 0u) bl.add_range(x, y);
          else if (selector == 1u) { bl.add_xor(x, y); /* could assert xy correctness on device if needed */ }
        }
        (void)xy;
      }
    }
  }
}

extern "C" int _apc_apply_bus(
  const Fp* d_output,
  size_t output_height,
  const DevInteraction* d_interactions,
  size_t n_interactions,
  const DevArgSpan* d_arg_spans,
  size_t n_arg_spans,
  const uint32_t* d_bytecode,
  size_t bytecode_len,
  int num_apc_calls,
  uint32_t var_range_bus_id,
  uint32_t tuple2_bus_id,
  uint32_t bitwise_bus_id,
  uint32_t* d_var_hist,
  size_t var_num_bins,
  uint32_t* d_tuple2_hist,
  uint32_t tuple2_sz0,
  uint32_t tuple2_sz1,
  uint32_t* d_bitwise_hist,
  uint32_t bitwise_num_bits
) {
  const int block_x = 128; // 4 warps
  const dim3 block(block_x, 1, 1);
  unsigned g = (unsigned)((n_interactions + 3) / 4);
  if (g == 0u) g = 1u;
  const dim3 grid(g, 1, 1); // each warp processes an interaction

  apc_apply_bus_kernel<<<grid, block>>>(
    d_output, output_height,
    d_interactions, n_interactions,
    d_arg_spans, n_arg_spans,
    d_bytecode, bytecode_len,
    num_apc_calls,
    var_range_bus_id,
    tuple2_bus_id,
    bitwise_bus_id,
    d_var_hist,
    var_num_bins,
    d_tuple2_hist,
    tuple2_sz0,
    tuple2_sz1,
    d_bitwise_hist,
    bitwise_num_bits
  );
  return (int)hipGetLastError();
}
