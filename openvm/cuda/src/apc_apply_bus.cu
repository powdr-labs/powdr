#include "hip/hip_runtime.h"
#include <stdint.h>
#include <assert.h>
#include <stdio.h>
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"
#include "primitives/histogram.cuh"

extern "C" {
  typedef struct {
    uint32_t bus_id; // Bus id this interaction targets (matches periphery chip bus id)
    uint32_t num_args; // Number of argument expressions for this interaction
    uint32_t args_index_off; // Starting index into the `DevArgSpan` array for this interaction's args. Layout: [mult, arg0, arg1, ...]
  } DevInteraction;

  typedef struct {
    uint32_t off; // Offset (in u32 words) into `bytecode` where this arg expression starts
    uint32_t len; // Length (instruction count) of this arg expression
  } DevArgSpan;
}

enum OpCode : uint32_t { 
  OP_PUSH_APC = 0, // Push the APC value onto the stack. Must be followed by the index of the value in the APC device buffer.
  OP_PUSH_CONST = 1, // Push a constant value onto the stack. Must be followed by the constant value.
  OP_ADD = 2, // Add the top two values on the stack.
  OP_SUB = 3, // Subtract the top two values on the stack.
  OP_MUL = 4, // Multiply the top two values on the stack.
  OP_NEG = 5, // Negate the top value on the stack.
};

// Fixed number of bits for bitwise lookup
static constexpr uint32_t BITWISE_NUM_BITS = 8u;
static constexpr int STACK_CAPACITY = 16;

// Inline helpers to safely manipulate the evaluation stack (capacity 16)
__device__ __forceinline__ void stack_push(Fp* stack, int& sp, Fp value) {
  assert(sp < STACK_CAPACITY && "Stack overflow");
  stack[sp++] = value;
}

__device__ __forceinline__ Fp stack_pop(Fp* stack, int& sp) {
  assert(sp > 0 && "Stack underflow");
  return stack[--sp];
}

__device__ __forceinline__ Fp eval_expr(const uint32_t* expr, uint32_t len,
                                        const Fp* __restrict__ apc_trace,
                                        size_t H, size_t r) {
  Fp stack[STACK_CAPACITY]; int sp = 0;
  for (uint32_t ip = 0; ip < len; ) {
    uint32_t op = expr[ip++];
    switch (op) {
      case OP_PUSH_APC: {
        uint32_t base = expr[ip++];
        stack_push(stack, sp, apc_trace[base + r]);
        break;
      }
      case OP_PUSH_CONST: {
        uint32_t u = expr[ip++];
        stack_push(stack, sp, Fp(u));
        break;
      }
      case OP_ADD: { Fp b = stack_pop(stack, sp); Fp a = stack_pop(stack, sp); stack_push(stack, sp, a + b); break; }
      case OP_SUB: { Fp b = stack_pop(stack, sp); Fp a = stack_pop(stack, sp); stack_push(stack, sp, a - b); break; }
      case OP_MUL: { Fp b = stack_pop(stack, sp); Fp a = stack_pop(stack, sp); stack_push(stack, sp, a * b); break; }
      case OP_NEG: { Fp a = stack_pop(stack, sp); stack_push(stack, sp, -a); break; }
    }
  }
  return stack[sp - 1];
}

__device__ __forceinline__ Fp eval_arg(
  const DevArgSpan& span,
  const uint32_t* __restrict__ d_bytecode,
  const Fp* __restrict__ apc_trace,
  size_t H,
  size_t r
) {
  return eval_expr(d_bytecode + span.off, span.len, apc_trace, H, r);
}

// Applies bus interactions to periphery histograms for a batch of APC rows
__global__ void apc_apply_bus_kernel(
  // APC related
  const Fp* __restrict__ d_output, // APC trace (column-major)
  size_t H, // APC trace height (rows)
  int num_apc_calls, // number of APC calls (rows)

  // Interaction related
  const uint32_t* __restrict__ d_bytecode, // bytecode for stack-machine expressions
  size_t bc_len, // bytecode length (u32 words)
  const DevInteraction* __restrict__ d_interactions, // interactions array
  size_t n_interactions, // number of interactions
  const DevArgSpan* __restrict__ d_arg_spans, // argument spans array
  size_t n_arg_spans, // number of arg spans

  // Variable range checker related
  uint32_t var_range_bus_id, // variable range checker bus id
  uint32_t* __restrict__ d_var_hist, // variable range histogram buffer
  size_t var_num_bins, // variable range histogram bin count

  // Tuple range checker related
  uint32_t tuple2_bus_id, // 2-tuple range checker bus id
  uint32_t* __restrict__ d_tuple2_hist, // tuple2 histogram buffer
  uint32_t tuple2_sz0, // tuple2 size dim0
  uint32_t tuple2_sz1, // tuple2 size dim1

  // Bitwise related
  uint32_t bitwise_bus_id, // bitwise lookup bus id
  uint32_t* __restrict__ d_bitwise_hist // bitwise lookup histogram buffer
) {
  const int warp = (threadIdx.x >> 5);
  const int lane = (threadIdx.x & 31);
  const int warps_per_block = (blockDim.x >> 5);

  // Each block processes a bus interaction and each thread within a block process an apc call, which evaluates multiple expressions.
  // TODO: can we parallelize over expression evaluation?
  for (int base = blockIdx.x * warps_per_block; base < (int)n_interactions; base += gridDim.x * warps_per_block) {
    int i = base + warp;
    if (i >= (int)n_interactions) return;
    DevInteraction intr = d_interactions[i];

    for (int r = lane; r < num_apc_calls; r += 32) {
      // multiplicity is stored as the first DevArgSpan for this interaction
      DevArgSpan mult_span = d_arg_spans[intr.args_index_off + 0];
      Fp mult = eval_arg(mult_span, d_bytecode, d_output, H, (size_t)r);
      // Evaluate args and apply based on bus id
      if (intr.bus_id == var_range_bus_id) {
        // expect [value, max_bits]
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 1];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 2];
        Fp v_fp = eval_arg(s0, d_bytecode, d_output, H, (size_t)r);
        Fp b_fp = eval_arg(s1, d_bytecode, d_output, H, (size_t)r);
        
        // histogram `num_bins` and index calculation depend on the `VariableRangeCheckerChipGPU` implementation
        uint32_t value = v_fp.asUInt32();
        uint32_t max_bits = b_fp.asUInt32();
        lookup::Histogram hist(d_var_hist, (uint32_t)var_num_bins);
        uint32_t idx = (1u << max_bits) + value; // `max_bit` 

        // apply multiplicity by looping; warp-level dedup in Histogram minimizes contention
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) hist.add_count(idx);
      } else if (intr.bus_id == tuple2_bus_id) {
        // expect [v0, v1]
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 1];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 2];
        Fp v0_fp = eval_arg(s0, d_bytecode, d_output, H, (size_t)r);
        Fp v1_fp = eval_arg(s1, d_bytecode, d_output, H, (size_t)r);
        
        // histogram `num_bins` and index calculation depend on the `RangeTupleCheckerChipGpu<2>` implementation
        uint32_t v0 = v0_fp.asUInt32();
        uint32_t v1 = v1_fp.asUInt32();
        lookup::Histogram hist(d_tuple2_hist, tuple2_sz0 * tuple2_sz1);
        uint32_t idx = v0 * tuple2_sz1 + v1;
        
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) hist.add_count(idx);
      } else if (intr.bus_id == bitwise_bus_id) {
        // expect [x, y, x_xor_y, selector]; we only update histogram if selector==range(0) or xor(1)
        DevArgSpan s0 = d_arg_spans[intr.args_index_off + 1];
        DevArgSpan s1 = d_arg_spans[intr.args_index_off + 2];
        DevArgSpan s2 = d_arg_spans[intr.args_index_off + 3];
        DevArgSpan s3 = d_arg_spans[intr.args_index_off + 4];
        Fp x_fp = eval_arg(s0, d_bytecode, d_output, H, (size_t)r);
        Fp y_fp = eval_arg(s1, d_bytecode, d_output, H, (size_t)r);
        Fp xy_fp = eval_arg(s2, d_bytecode, d_output, H, (size_t)r);
        Fp sel_fp = eval_arg(s3, d_bytecode, d_output, H, (size_t)r);

        uint32_t x = x_fp.asUInt32();
        uint32_t y = y_fp.asUInt32();
        uint32_t xy = xy_fp.asUInt32();
        uint32_t selector = sel_fp.asUInt32();
        BitwiseOperationLookup bl(d_bitwise_hist, BITWISE_NUM_BITS);
        
        for (uint32_t k = 0; k < (uint32_t)mult.asUInt32(); ++k) {
          if (selector == 0u) bl.add_range(x, y);
          else if (selector == 1u) { bl.add_xor(x, y); /* could assert xy correctness on device if needed */ }
          else { assert(false && "Invalid selector"); }
        }
        (void)xy;
      }
    }
  }
}

// Host entry point to launch the kernel that applies bus interactions
extern "C" int _apc_apply_bus(
  // APC related
  const Fp* d_output, // APC trace (column-major), device pointer
  size_t output_height, // APC trace height (rows)
  int num_apc_calls, // number of APC calls (rows)

  // Interaction related
  const uint32_t* d_bytecode, // bytecode buffer (device)
  size_t bytecode_len, // length of bytecode (u32 words)
  const DevInteraction* d_interactions, // interactions array (device)
  size_t n_interactions, // number of interactions
  const DevArgSpan* d_arg_spans, // argument spans (device)
  size_t n_arg_spans, // number of arg spans

  // Variable range checker related
  uint32_t var_range_bus_id, // variable range checker bus id
  uint32_t* d_var_hist, // variable range histogram (device)
  size_t var_num_bins, // number of bins in variable range histogram

  // Tuple range checker related
  uint32_t tuple2_bus_id, // 2-tuple range checker bus id
  uint32_t* d_tuple2_hist, // tuple2 histogram (device)
  uint32_t tuple2_sz0, // tuple2 size dim0
  uint32_t tuple2_sz1, // tuple2 size dim1

  // Bitwise related
  uint32_t bitwise_bus_id, // bitwise lookup bus id
  uint32_t* d_bitwise_hist // bitwise lookup histogram (device)
) {
  const int block_x = 128; // 4 warps
  const dim3 block(block_x, 1, 1);
  unsigned g = (unsigned)((n_interactions + 3) / 4);
  if (g == 0u) g = 1u;
  const dim3 grid(g, 1, 1); // each warp processes an interaction

  apc_apply_bus_kernel<<<grid, block>>>(
    // APC related
    d_output, output_height, num_apc_calls,

    // Interaction related
    d_bytecode, bytecode_len, d_interactions, n_interactions, d_arg_spans, n_arg_spans,

    // Variable range checker related
    var_range_bus_id, d_var_hist, var_num_bins,

    // Tuple range checker related
    tuple2_bus_id, d_tuple2_hist, tuple2_sz0, tuple2_sz1,

    // Bitwise related
    bitwise_bus_id, d_bitwise_hist
  );
  return (int)hipGetLastError();
}
