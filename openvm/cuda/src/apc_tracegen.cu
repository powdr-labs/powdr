#include "hip/hip_runtime.h"
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"

// ============================================================================================
// Types
// ============================================================================================

struct OriginalAir {
    int width;               // number of columns
    int height;              // number of rows (Ha)
    const Fp* buffer;        // column-major base: col*height + row
    int row_block_size;      // stride between used rows
    int substitutions_offset;// offset in d_subs
    int substitutions_length;// count in d_subs for this AIR
};

struct Subst {
    int col;      // source column within this AIR
    int row;      // base row offset within the row-block
    int apc_col;  // destination APC column
};

// ============================================================================================
// Kernel: one block per OriginalAir; each warp handles one substitution (APC column).
// ============================================================================================

__global__ void apc_tracegen_kernel(
    Fp* __restrict__ d_output,                         // [output_height * output_width], column-major
    const OriginalAir* __restrict__ d_original_airs,   // metadata per AIR
    const Subst* __restrict__ d_subs,                  // all substitutions
    size_t output_height,                              // H_out
    int num_apc_calls                                  // number of APC calls
) {
    const int air_id = blockIdx.x;
    const OriginalAir air = d_original_airs[air_id];

    const Fp* __restrict__ src_base = air.buffer;
    const int Ha  = air.height;
    const int RBS = air.row_block_size;

    const int lane  = threadIdx.x & 31;     // 0..31
    const int warp  = threadIdx.x >> 5;     // warp index in block
    const int warps_per_block = blockDim.x >> 5;

    // Process this AIR's substitutions in batches of warps_per_block
    for (int base = 0; base < air.substitutions_length; base += warps_per_block) {
        const int rel = base + warp;
        if (rel >= air.substitutions_length) break;

        const Subst sub = d_subs[air.substitutions_offset + rel];

        // Column bases (column-major)
        const size_t dst_col_base = (size_t)sub.apc_col * (size_t)output_height;
        const size_t src_col_base = (size_t)sub.col     * (size_t)Ha;

        // Each lane writes rows lane, lane+32, lane+64, ... (coalesced per warp)
        for (size_t r = (size_t)lane; r < num_apc_calls; r += 32) {
            const size_t src_r = (size_t)sub.row + r * (size_t)RBS;
            if (src_r < (size_t)Ha) {
                d_output[dst_col_base + r] = src_base[src_col_base + src_r];
            }
        }
        // Warps are independent for different substitutions; no syncthreads needed here.
    }
}

// ============================================================================================
// Host launcher wrapper — callable from Rust FFI or cudarc
// ============================================================================================

extern "C" int _apc_tracegen(
    Fp*                      d_output,          // [output_height * output_width], column-major
    size_t                   output_height,     // H_out
    const OriginalAir*       d_original_airs,   // device array, length = n_airs
    size_t                   n_airs,            // one block per AIR
    const Subst*             d_subs,            // device array of all substitutions
    int                      num_apc_calls      // number of APC calls
) {
    assert((output_height & (output_height - 1)) == 0);  // power-of-two height check

    const int block_x = 32;
    const dim3 block(block_x, 1, 1);
    const dim3 grid((unsigned int)n_airs, 1, 1);

    apc_tracegen_kernel<<<grid, block>>>(
        d_output, d_original_airs, d_subs, output_height, num_apc_calls
    );
    return (int)hipGetLastError();
}