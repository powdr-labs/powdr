#include "hip/hip_runtime.h"
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"
#include "expr_eval.cuh"

// ============================================================================================
// Types
// ============================================================================================

struct OriginalAir {
    int width;               // number of columns
    int height;              // number of rows (Ha)
    const Fp* buffer;        // column-major base: col*height + row
    int row_block_size;      // stride between used rows
    int substitutions_offset;// offset in d_subs
    int substitutions_length;// count in d_subs for this AIR
};

struct Subst {
    int col;      // source column within this AIR
    int row;      // base row offset within the row-block
    int apc_col;  // destination APC column
};

struct DerivedColumn {
    int index; // destination APC column index
    Fp  value; // constant value for valid rows
};

extern "C" {
  typedef struct {
    uint64_t col_base; // precomputed destination base offset = apc_col_index * H
    DevArgSpan span;   // expression span encoding this column's value
  } DerivedExprSpec;
}

// ============================================================================================
// Kernel: one block per OriginalAir; each warp handles one substitution (APC column).
// ============================================================================================

__global__ void apc_tracegen_kernel(
    Fp* __restrict__ d_output,                         // [output_height * output_width], column-major
    const OriginalAir* __restrict__ d_original_airs,   // metadata per AIR
    const Subst* __restrict__ d_subs,                  // all substitutions
    size_t output_height,                              // H_out
    int num_apc_calls                                  // number of APC calls
) {
    const int air_id = blockIdx.x;
    const OriginalAir air = d_original_airs[air_id];

    const Fp* __restrict__ src_base = air.buffer;
    const int Ha  = air.height;
    const int RBS = air.row_block_size;

    const int lane  = threadIdx.x & 31;     // 0..31
    const int warp  = threadIdx.x >> 5;     // warp index in block
    const int warps_per_block = blockDim.x >> 5;

    // Process this AIR's substitutions in batches of warps_per_block
    for (int base = 0; base < air.substitutions_length; base += warps_per_block) {
        const int rel = base + warp;
        if (rel >= air.substitutions_length) break;

        const Subst sub = d_subs[air.substitutions_offset + rel];

        // Column bases (column-major)
        const size_t dst_col_base = (size_t)sub.apc_col * (size_t)output_height;
        const size_t src_col_base = (size_t)sub.col     * (size_t)Ha;

        // Each lane writes rows lane, lane+32, lane+64, ... (coalesced per warp)
        // Loop over full output height; zero-pad rows beyond `num_apc_calls`.
        for (size_t r = (size_t)lane; r < (size_t)output_height; r += 32) {
            if (r < (size_t)num_apc_calls) {
                const size_t src_r = (size_t)sub.row + r * (size_t)RBS;
                if (src_r < (size_t)Ha) {
                    d_output[dst_col_base + r] = src_base[src_col_base + src_r];
                }
            } else {
                d_output[dst_col_base + r] = Fp(0);
            }
        }
        // Warps are independent for different substitutions; no syncthreads needed here.
    }
}

// ============================================================================================
// Derived expressions: lane-per-row evaluator, sequential over derived columns per row
// ============================================================================================

__global__ void apc_apply_derived_expr_kernel(
    Fp* __restrict__ d_output,   // APC trace (column-major)
    size_t H,                    // rows (height)
    int num_apc_calls,           // number of valid rows
    const DerivedExprSpec* __restrict__ d_specs, // derived expression specs
    size_t n_cols,               // number of derived columns
    const uint32_t* __restrict__ d_bytecode // shared bytecode buffer
) {
    const size_t total_threads = (size_t)gridDim.x * (size_t)blockDim.x;
    const size_t tid = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;

    for (size_t r = tid; r < H; r += total_threads) {
        if (r < (size_t)num_apc_calls) {
            // Compute and write each derived column for this row
            for (size_t i = 0; i < n_cols; ++i) {
                const DerivedExprSpec spec = d_specs[i];
                const size_t col_base = (size_t)spec.col_base;
                const Fp v = eval_arg(spec.span, d_bytecode, d_output, H, r);
                d_output[col_base + r] = v;
            }
        } else {
            // Zero-fill non-APC rows
            for (size_t i = 0; i < n_cols; ++i) {
                const size_t col_base = (size_t)d_specs[i].col_base;
                d_output[col_base + r] = Fp(0);
            }
        }
    }
}

extern "C" int _apc_apply_derived_expr(
    Fp*                d_output,
    size_t             output_height,
    int                num_apc_calls,
    const DerivedExprSpec* d_specs,
    size_t             n_cols,
    const uint32_t*    d_bytecode
) {
    if (n_cols == 0) return 0;
    const int block_x = 256; // more lanes to cover rows
    const dim3 block(block_x, 1, 1);
    unsigned g = (unsigned)((output_height + block_x - 1) / block_x);
    if (g == 0u) g = 1u;
    const dim3 grid(g, 1, 1);
    apc_apply_derived_expr_kernel<<<grid, block>>>(
        d_output, output_height, num_apc_calls, d_specs, n_cols, d_bytecode
    );
    return (int)hipGetLastError();
}

// ============================================================================================
// Host launcher wrapper — callable from Rust FFI or cudarc
// ============================================================================================

extern "C" int _apc_tracegen(
    Fp*                      d_output,          // [output_height * output_width], column-major
    size_t                   output_height,     // H_out
    const OriginalAir*       d_original_airs,   // device array, length = n_airs
    size_t                   n_airs,            // one block per AIR
    const Subst*             d_subs,            // device array of all substitutions
    int                      num_apc_calls      // number of APC calls
) {
    assert((output_height & (output_height - 1)) == 0);  // power-of-two height check

    const int block_x = 32;
    const dim3 block(block_x, 1, 1);
    const dim3 grid((unsigned int)n_airs, 1, 1);

    apc_tracegen_kernel<<<grid, block>>>(
        d_output, d_original_airs, d_subs, output_height, num_apc_calls
    );
    return (int)hipGetLastError();
}