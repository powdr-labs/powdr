#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "primitives/histogram.cuh"  // Histogram (in lookup::), RangeTupleChecker (global)

template <uint32_t N>
__global__ void rt_add_count_kernel(uint32_t* global_hist,
                                    const uint32_t* sizes,
                                    const uint32_t* values) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        uint32_t sz[N];
        #pragma unroll
        for (uint32_t i = 0; i < N; ++i) sz[i] = sizes[i];

        // RangeTupleChecker is in the global namespace (not lookup::)
        RangeTupleChecker<N> rtc(global_hist, sz);

        uint32_t vals[N];
        #pragma unroll
        for (uint32_t i = 0; i < N; ++i) vals[i] = values[i];

        rtc.add_count(vals);
    }
}

extern "C" int _range_tuple2_add_count(uint32_t* d_hist,
                                       const uint32_t* sizes2,
                                       const uint32_t* values2) {
    rt_add_count_kernel<2><<<1, 1>>>(d_hist, sizes2, values2);
    return hipGetLastError();
}