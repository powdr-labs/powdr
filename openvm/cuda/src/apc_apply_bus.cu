#include "hip/hip_runtime.h"
#include <stdint.h>
#include "primitives/buffer_view.cuh"
#include "primitives/constants.h"
#include "primitives/trace_access.h"

extern "C" {
typedef struct {
  uint32_t id;
  uint32_t num_args;
  uint32_t mult_off;
  uint32_t mult_len;
  uint32_t args_index_off;
} DevInteraction;

typedef struct {
  uint32_t off;
  uint32_t len;
} DevArgSpan;
}

enum OpCode : uint32_t { OP_PUSH_APC = 0, OP_PUSH_CONST = 1, OP_ADD = 2, OP_SUB = 3, OP_MUL = 4, OP_NEG = 5 };

__device__ __forceinline__ Fp eval_prog(const uint32_t* prog, uint32_t len,
                                        const Fp* __restrict__ col_major,
                                        size_t H, size_t r) {
  Fp stack[16]; int sp = 0; // not sure if enough or what's the typical stack size for our expressions
  for (uint32_t ip = 0; ip < len; ) {
    uint32_t op = prog[ip++];
    switch (op) {
      case OP_PUSH_APC: {
        uint32_t base = prog[ip++];
        stack[sp++] = col_major[base + r];
        break;
      }
      case OP_PUSH_CONST: {
        uint32_t u = prog[ip++];
        stack[sp++] = Fp(u);
        break;
      }
      case OP_ADD: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a + b; break; }
      case OP_SUB: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a - b; break; }
      case OP_MUL: { Fp b = stack[--sp]; Fp a = stack[--sp]; stack[sp++] = a * b; break; }
      case OP_NEG: { Fp a = stack[--sp]; stack[sp++] = -a; break; }
    }
  }
  return stack[sp - 1];
}

__global__ void apc_apply_bus_kernel(
  const Fp* __restrict__ d_output, size_t H,
  const DevInteraction* __restrict__ d_interactions, size_t n_interactions,
  const DevArgSpan* __restrict__ d_arg_spans, size_t n_arg_spans,
  const uint32_t* __restrict__ d_bytecode, size_t bc_len,
  int num_apc_calls
) {
  const int warp = (threadIdx.x >> 5);
  const int lane = (threadIdx.x & 31);
  const int warps_per_block = (blockDim.x >> 5);
  for (int base = blockIdx.x * warps_per_block; base < (int)n_interactions; base += gridDim.x * warps_per_block) {
    int i = base + warp;
    if (i >= (int)n_interactions) return;
    DevInteraction intr = d_interactions[i];

    for (int r = lane; r < num_apc_calls; r += 32) {
      const uint32_t* mult_prog = d_bytecode + intr.mult_off;
      Fp mult = eval_prog(mult_prog, intr.mult_len, d_output, H, (size_t)r);
      // Evaluate args
      // Note: args are described by spans starting at args_index_off, length num_args
      for (uint32_t j = 0; j < intr.num_args; ++j) {
        DevArgSpan span = d_arg_spans[intr.args_index_off + j];
        const uint32_t* arg_prog = d_bytecode + span.off;
        Fp arg_val = eval_prog(arg_prog, span.len, d_output, H, (size_t)r);
        // TODO: store/apply (mult, arg_val) according to bus id intr.id
        (void)arg_val;
      }
      (void)mult;
    }
  }
}

extern "C" int _apc_apply_bus(
  const Fp* d_output,
  size_t output_height,
  const DevInteraction* d_interactions,
  size_t n_interactions,
  const DevArgSpan* d_arg_spans,
  size_t n_arg_spans,
  const uint32_t* d_bytecode,
  size_t bytecode_len,
  int num_apc_calls
) {
  const int block_x = 128; // 4 warps
  const dim3 block(block_x, 1, 1);
  const dim3 grid(max(1u, (unsigned)((n_interactions + 3) / 4)), 1, 1); // each warp processes an interaction

  apc_apply_bus_kernel<<<grid, block>>>(
    d_output, output_height,
    d_interactions, n_interactions,
    d_arg_spans, n_arg_spans,
    d_bytecode, bytecode_len,
    num_apc_calls
  );
  return (int)hipGetLastError();
}
